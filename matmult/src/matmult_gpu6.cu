#include <hip/hip_runtime.h>

__global__ void matmult_gpu6Kernel(int m, int n, int k, double * d_A, double * d_B, double * d_C);


extern "C" {
void matmult_gpu6(int m, int n, int k, double * A, double * B, double * C){
	double * d_A, * d_B, * d_C;

	hipMalloc((void **)&d_A, m * k * sizeof(double *));
	hipMalloc((void **)&d_B, k * n * sizeof(double *));
	hipMalloc((void **)&d_C, m * n * sizeof(double *));

	hipMemcpy(d_A, A, m * k * sizeof(double *), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(double *), hipMemcpyHostToDevice);

	//kernel block and grid size
    dim3 dimBlock(16,16,4);
    dim3 dimGrid((int)ceil(((double)m)/16), (int)ceil(((double)n)/16),(int)ceil(((double)k)/4));  

    matmult_gpu6Kernel<<<dimGrid,dimBlock>>>(m, n, k, d_A, d_B, d_C);

    hipMemcpy(C, d_C, m * n * sizeof(double *), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
}

__global__ void matmult_gpu6Kernel(int m, int n, int k, double * d_A, double * d_B, double * d_C){

    int i, j, l;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    j = blockIdx.y * blockDim.y + threadIdx.y;
	l = blockIdx.z * blockDim.z + threadIdx.z;

	if(i < m && j < n && l < k){
		atomicAdd(&d_C[i*n + j],d_A[i*k + l] * d_B[l*n + j]);
	}

}


