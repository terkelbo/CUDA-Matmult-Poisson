#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C" {
void matmult_gpulib(int m, int n, int k, double * A, double * B, double * C){
   	hipblasHandle_t handle;
  	hipblasCreate(&handle);

  	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	double * d_A, * d_B, * d_C;

	hipMalloc((void **)&d_A, m * k * sizeof(double *));
	hipMalloc((void **)&d_B, k * n * sizeof(double *));
	hipMalloc((void **)&d_C, m * n * sizeof(double *));

	hipMemcpy(d_A, A, m * k * sizeof(double *), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(double *), hipMemcpyHostToDevice);

	hipblasDgemm(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
               m, n, k,
               alpha,
               d_A, m,
               d_B, k,
               beta,
               d_C, m);

    hipMemcpy(C, d_C, m * n * sizeof(double *), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);
}
}
