#include <hip/hip_runtime.h>

__global__ void matmult_gpu1Kernel(int m, int n, int k, double * d_A, double * d_B, double * d_C);

extern "C" {
void matmult_gpu1(int m, int n, int k, double * A, double * B, double * C){
	double * d_A, * d_B, * d_C;

	hipMalloc((void **)&d_A, m * k * sizeof(double *));
	hipMalloc((void **)&d_B, k * n * sizeof(double *));
	hipMalloc((void **)&d_C, m * n * sizeof(double *));

	hipMemcpy(d_A, A, m * k * sizeof(double *), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(double *), hipMemcpyHostToDevice);

    matmult_gpu1Kernel<<<1,1>>>(m, n, k, d_A, d_B, d_C);

    hipMemcpy(C, d_C, m * n * sizeof(double *), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
}

__global__ void matmult_gpu1Kernel(int m, int n, int k, double * d_A, double * d_B, double * d_C){

    int i, j, l;
    double x;
	
	
	for(i=0;i < m; i++){
		for(j=0;j<n;j++){
			d_C[i*n + j]=0;
		}
		for(l=0;l < k;l++){
			x = d_A[i*k + l];
			for(j=0;j < n; j++){
				d_C[i*n + j] += x * d_B[l*n + j];
			}
		}
	}

}
