#include "hip/hip_runtime.h"
#include "jacobi_gpu1.h"

__global__ void jacobi(int n, double h, double * u_old, double * u_new, double * f){

	int i, j;

	for(i = 1; i < (n + 1); i++){
		for(j = 1; j < (n + 1); j++){
			u_new[i*(n + 2) + j] = 0.25*(u_old[(i-1)*(n + 2) + j] + u_old[(i+1)*(n + 2) + j] + u_old[i*(n + 2) + j-1] + u_old[i*(n + 2) + j+1] + h*h*f[i*(n + 2) + j]);
		}
	}
}
