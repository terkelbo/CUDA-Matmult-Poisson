#include "hip/hip_runtime.h"
#include "jacobi_gpu2.h"

__global__ void jacobi(int n, double h, double * u_old, double * u_new, double * f){


	int i = blockIdx.x * blockDim.x + threadIdx.x;  
	int j = blockIdx.y * blockDim.y + threadIdx.y;  

	if(i > 0 && i < (n+1) && j > 0 && j < (n+1)){
		u_new[i*(n + 2) + j] = 0.25*(u_old[(i-1)*(n + 2) + j] + u_old[(i+1)*(n + 2) + j] + u_old[i*(n + 2) + j-1] + u_old[i*(n + 2) + j+1] + h*h*f[i*(n + 2) + j]);
	}
	
		
	
}
