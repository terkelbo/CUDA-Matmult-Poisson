extern "C" { 

#include <hip/hip_runtime.h>
#include "cblas.h" 

void matmult_lib(int m, int n , int k , double *A, double *B, double *C){
	int alpha = 1, beta = 0;

	cblas_dgemm(CblasRowMajor,CblasNoTrans,CblasNoTrans,m,n,k,alpha,A,k,B,n,beta,C,n);
}
}
